#include <windows.h>
#include <cinttypes>
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <vector>

typedef const char* const* const const_char_array_t;
enum class RETURN_CODE
{
	SUCCESS =  0,
	FAILURE = -1,
	SEH		=  2,
};
enum class COPY_DIRECTION
{
	TO_GPU = hipMemcpyHostToDevice,
	TO_CPU = hipMemcpyDeviceToHost,
};

/*
	GPU memory wrapper.
	Ownership of given buffer is taken.
	The host buffer would not be available for use after
	instantiation of the class.
	The caller of the Get method is responsible to free
	the allocated host buffer.
*/
template<typename T>
class GPUMemory
{
public:
	GPUMemory(const T* buffer, const std::size_t size) :
		m_DeviceBuffer(nullptr),
		m_BufferSize(size)
	{
		// Allocating device buffer on the GPU.
		const auto allocationResult = hipMalloc<T>((T**)&m_DeviceBuffer, size);
		
		// Copy data from host to device GPU.
		const auto memcpyResult = hipMemcpy(
			(void*)m_DeviceBuffer, 
			(void*)buffer, 
			size, 
			static_cast<hipMemcpyKind>(COPY_DIRECTION::TO_GPU));
		
		// Remove data from host.
		ZeroMemory((void*)buffer, size);
	}

	T* Get(void)
	{
		T* ptr = new T[m_BufferSize];
		const auto memcpyResult = hipMemcpy(
			(void*)ptr,
			(void*)m_DeviceBuffer,
			m_BufferSize,
			static_cast<hipMemcpyKind>(COPY_DIRECTION::TO_CPU));
		return ptr;
	}

	~GPUMemory() noexcept
	{
		hipFree(m_DeviceBuffer);
	}

private:
	T* m_DeviceBuffer = nullptr;
	std::size_t m_BufferSize = 0;
};

__host__ int main(const std::uint8_t argc, const_char_array_t argv)
{
	UNREFERENCED_PARAMETER(argc);
	UNREFERENCED_PARAMETER(argv);

	std::vector<int> buffer = { 0xff };

	// Initiate host buffer with some data.
	const std::uint8_t hostBuffer[] =
	{0xde, 0xad, 0xbe, 0xef,
	 0xde, 0xad, 0xbe, 0xef,
	 0xde, 0xad, 0xbe, 0xef,
	 0xde, 0xad, 0xbe, 0xef,
	 0xde, 0xad, 0xbe, 0xef};

	// Send buffer to GPU for later use.
	GPUMemory<std::uint8_t> gpuBuffer(hostBuffer, sizeof(hostBuffer));

	/*
	...
	hostBuffer is not available here.
	Can not be found in process memory dump.
	...
	*/

	// Use the hidden GPU memory only when required.
	const auto hiddenBuffer = gpuBuffer.Get();

	delete hiddenBuffer;
	return static_cast<int>(RETURN_CODE::SUCCESS);
}